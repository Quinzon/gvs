#include "hip/hip_runtime.h"
﻿#include <torch/extension.h>
#include <stdexcept>
#include <sstream>


__global__ void cudaLinear(float* X, float* W, float* B, float* Y, int M, int N, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M)
    {
        for (int j = 0; j < K; j++)
        {
            float sum = 0.0f;

            for (int k = 0; k < N; k++)
            {
                //sum += X[row * N + k] * W[k * K + j]; //Условимся, что матрица W уже транспонирована
				sum += X[row * N + k] * W[j * N + k];
            }

            Y[row * K + j] = sum + B[j];
        }
    }
}

// Функция для вызова из Python
void gpu_linear(torch::Tensor input, torch::Tensor weights, torch::Tensor bias, torch::Tensor output)
{

	int Mx = input.size(0);   //строки X    
    int Nx = input.size(1);   //столбики X
    
    //кол-во столбцов должно быть равно кол-ву строк
    //Nx и Nw должны быть равны, так как матрица W пока не транспонирована
    //Mw и Mb должны быть равны, так как матрица W пока не транспонирована

    int Mw = weights.size(0);   //строки W
    int Nw = weights.size(1);   //столбики W

	int Mb = bias.size(0);   //размер вектора 

    int threadsPerBlock = 256;
    int blocksPerGrid = (Mx + threadsPerBlock - 1) / threadsPerBlock;	
	
	//Проверка размеров тензоров
    if (Nx != Nw)
	{
        std::stringstream ss;
        ss << "First matrix columns (" << Nx << ") must match number of columns in second matrix (" << Nw << ")";
        throw std::runtime_error(ss.str());
    }
	
    if (Mw != Mb)
	{
        std::stringstream ss;
        ss << "Second matrix rows (" << Mw << ") must match size of a vector (" << Mb << ")";
        throw std::runtime_error(ss.str());
    }
	
    if (Mw != output.size(1))
	{
        std::stringstream ss;
        ss << "Output matrix columns (" << output.size(1) << ") must match number of rows in second matrix (" << Mw << ")";
        throw std::runtime_error(ss.str());
    }

	cudaLinear<<<blocksPerGrid, threadsPerBlock>>>(input.data_ptr<float>(), weights.data_ptr<float>(), bias.data_ptr<float>(), output.data_ptr<float>(), Mx, Nx, Mw);

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		std::stringstream ss;
		ss << "CUDA error: " << hipGetErrorString(err);
		throw std::runtime_error(ss.str());
	}
	
}

PYBIND11_MODULE(kernel_cuda_extension, m)
{
    m.def("gpu_linear", &gpu_linear, "Linear Layer Calculation");
}