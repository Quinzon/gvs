#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

__global__ void cudaLinear(float* X, float* W, float* B, float* Y, int M, int N, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M)
    {
        for (int j = 0; j < K; j++)
        {
            float sum = 0.0f;

            for (int k = 0; k < N; k++)
            {
                //sum += X[row * N + k] * W[k * K + j]; //Условимся, что матрица W уже транспонирована
                sum += X[row * N + k] * W[j * K + k];
            }

            Y[row * K + j] = sum + B[j];
        }
    }
}

void cudaLinear_helper(float* X, float* W, float* B, float* Y, int M, int N, int K)
{
    float* d_X, * d_W, * d_B, * d_Y;

    hipMalloc((void**)&d_X, M * N * sizeof(float));
    hipMalloc((void**)&d_W, N * K * sizeof(float));
    hipMalloc((void**)&d_B, K * sizeof(float));
    hipMalloc((void**)&d_Y, M * K * sizeof(float));


    hipMemcpy(d_X, X, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_W, W, N * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * sizeof(float), hipMemcpyHostToDevice);


    //int threadsPerBlock = 256;
    int blocksPerGrid = (M + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    cudaLinear <<< blocksPerGrid, THREADS_PER_BLOCK >>> (d_X, d_W, d_B, d_Y, M, N, K);

    hipMemcpy(Y, d_Y, M * K * sizeof(float), hipMemcpyDeviceToHost);

    //delete[] d_W_transposed;
    hipFree(d_X);
    hipFree(d_W);
    hipFree(d_B);
    hipFree(d_Y);
}
