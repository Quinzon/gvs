
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <ctime>
#include <hiprand.h>
#include "kernel.cuh"

#include <vector>
#include <cstdlib>
#include <ctime>

using namespace std;

void generateRandomValues(float* matrix, int rows, int cols)
{
    for (int i = 0; i < rows * cols; ++i)
    {
        matrix[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

//void matrixVectorAddCPU(const float* X, const float* W, const float* B, float* Y, int M, int N, int K)
//{
//    for (int i = 0; i < M; i++) 
//    {
//        for (int j = 0; j < K; j++) 
//        {
//            float sum = 0.0f;
//
//            for (int k = 0; k < N; k++) 
//            {
//                //sum += X[i * N + k] * W[k * K + j];   //���������, ��� ������� W ��� ���������������
//                sum += X[i * N + k] * W[j * K + k];
//            }
//
//            Y[i * K + j] = sum + B[j];
//        }
//    }
//}

//X, W - ������� �������� Mx �� Nx � Mw �� Nw ��������������
//B - ������ �������� Mb
//Y - �������������� ������� �������� Mx �� Nw
//Y = X * Wt + B - ������� W �� ���������������
//void matrixVectorAddCPU(const float* X, const float* W, const float* B, float* Y, int Mx, int Nx, int Mw, int Nw, int Mb)
//{
//    for (int i = 0; i < Mx; i++)
//    {
//        for (int j = 0; j < Mx; j++)
//        {
//            float sum = 0.0f;
//
//            for (int k = 0; k < Nw; k++)
//            {
//                //sum += X[i * N + k] * W[k * K + j];   //���������, ��� ������� W ��� ���������������
//                sum += X[i * Nw + k] * W[i * Mx + k];
//            }
//
//            Y[i * Mx + j] = sum + B[j];
//        }
//    }
//}

void matrixVectorAddCPU(const float* X, const float* W, const float* B, float* Y, int M, int N, int K)
{
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < K; j++)
        {
            float sum = 0.0f;

            for (int k = 0; k < N; k++)
            {
                sum += X[i * N + k] * W[j * K + k];
            }

            Y[i * K + j] = sum + B[j];
        }
    }
}

void printMatrix(const float* matrix, int rows, int cols)
{

    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            cout << matrix[i * cols + j] << " ";
        }
        cout << "\n";
    }
}

void transposeMatrixDevice(float* input, float* output, int rows, int cols)
{
    for (int i = 0; i < rows; ++i)
    {
        for (int j = 0; j < cols; ++j)
        {
            output[j * rows + i] = input[i * cols + j];
        }
    }
}

int main()
{

    setlocale(LC_ALL, "Russian");

    const int Mx = 3;   //������ X    
    const int Nx = 4;   //�������� X
    
    //���-�� �������� ������ ���� ����� ���-�� �����
    //Nx � Nw ������ ���� �����, ��� ��� ������� W ���� �� ���������������
    //Mw � Mb ������ ���� �����, ��� ��� ������� W ���� �� ���������������

    const int Mw = 4;   //������ W
    const int Nw = 4;   //�������� W

    const int Mb = 4;   //������ ������� 

    float h_X[Mx * Nx];
    float h_W[Mw * Nw];
    float h_B[Mb];

    //srand(static_cast<unsigned>(time(0)));
    srand(time(nullptr));

    generateRandomValues(h_X, Mx, Nx);

    generateRandomValues(h_W, Mw, Nw);

    generateRandomValues(h_B, 1, Mb);

    float h_Y_GPU[Mx * Mw];
    float h_Y_CPU[Mx * Mw];

    cout << "\n������� X:\n";
    printMatrix(h_X, Mx, Nx);

    cout << "\n������� W:\n";
    printMatrix(h_W, Mw, Nw);

    cout << "\n������ B:\n";
    printMatrix(h_B, 1, Mb);

    if ((Nx == Nw) && (Mw == Mb))
    {
        cudaLinear_helper(h_X, h_W, h_B, h_Y_GPU, Mx, Nx, Mw);

        matrixVectorAddCPU(h_X, h_W, h_B, h_Y_CPU, Mx, Nx, Mw);

        cout << "\n��������� GPU:\n";
        printMatrix(h_Y_GPU, Mx, Mw);

        cout << "\n��������� CPU:\n";
        printMatrix(h_Y_CPU, Mx, Mw);
    }
    else
    {
        cout << "\n������� �� ��������!\n";
    }

    return 0;
}